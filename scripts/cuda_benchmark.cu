#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

// Fehlerbehandlung Makro
#define CUDA_CHECK(call) do { \
cudaError_t err = call; \
if (err != hipSuccess) { \
    fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
    exit(1); \
} \
} while(0)

int main() {
    // 100 MB Test
    size_t size = 100 * 1024 * 1024;
    float *h_data = NULL;
    float *d_data = NULL;
    hipEvent_t start, stop;

    printf("CUDA Bandwidth Test - 100MB\n");
    printf("============================\n");

    // Device Info
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    printf("Device: %s\n\n", prop.name);

    // Memory allozieren
    CUDA_CHECK(hipHostAlloc((void**)&h_data, size, hipHostMallocDefault));
    CUDA_CHECK(hipMalloc((void**)&d_data, size));

    // Events erstellen
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Test data initialisieren
    for (int i = 0; i < size/4; i++) {
        h_data[i] = i % 1000;
    }

    printf("Running 10 iterations...\n\n");

    float total_write_time = 0.0f;
    float total_read_time = 0.0f;

    // 10 Tests
    for (int i = 0; i < 10; i++) {
        // Write test (Host->Device)
        CUDA_CHECK(hipEventRecord(start));
        CUDA_CHECK(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));

        float write_ms;
        CUDA_CHECK(hipEventElapsedTime(&write_ms, start, stop));

        // Read test (Device->Host)
        CUDA_CHECK(hipEventRecord(start));
        CUDA_CHECK(hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));

        float read_ms;
        CUDA_CHECK(hipEventElapsedTime(&read_ms, start, stop));

        total_write_time += write_ms;
        total_read_time += read_ms;

        // Bandbreite berechnen
        float write_bw = (100.0f / 1024.0f) / (write_ms / 1000.0f);  // GB/s
        float read_bw = (100.0f / 1024.0f) / (read_ms / 1000.0f);    // GB/s

        printf("Test %2d: Write %.2f GB/s, Read %.2f GB/s\n", i+1, write_bw, read_bw);
    }

    // Durchschnitt berechnen
    float avg_write_bw = (100.0f / 1024.0f) / ((total_write_time / 10.0f) / 1000.0f);
    float avg_read_bw = (100.0f / 1024.0f) / ((total_read_time / 10.0f) / 1000.0f);

    printf("\nAverage Results:\n");
    printf("Write Bandwidth: %.2f GB/s\n", avg_write_bw);
    printf("Read Bandwidth:  %.2f GB/s\n", avg_read_bw);

    // Cleanup
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipHostFree(h_data));

    return 0;
}
